#include "hip/hip_runtime.h"
/**
 * == Personal ==
 * Name: Christopher Lane
 * ID: 1435876
 *
 * == Goals ==
 * Block Scan: Achieved
 * Full Scan: Not Achieved
 * BCAO: Achieved
 *
 * == Times ==
 ∗ Block scan w/o BCAO: 0.00199 s
 ∗ Block scan w/ BCAO:  0.00214 s
 ∗ Full scan w/o BCAO:
 ∗ Full scan w/ BCAO:
 *
 * == Hardware ==
 * CPU: i7-4710MQ
 * GPU: GTX 860M
 *
 * == Comments ==
 * Implementation:
 * - Length of array is padded to the nearest power of 2 and the array is padded with 0s to that length.
 *
 * Performance:
 *
 */

#include <stdio.h>
#include <math.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#define NUM_BANKS 32
#define LOG_NUM_BANKS 5
#define CONFLICT_FREE_OFFSET(n) \
    ((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))

void refScan(int *h_output, int *h_input, const int len) {
    h_output[0] = 0;

    for (int i = 1; i < len; ++i) {
        h_output[i] = h_input[i - 1] + h_output[i - 1];
    }
}

extern __shared__ int temp[]; // allocated on invocation

__global__ void blockScan(int *d_Output, int *d_Input, int len) {
    int thid = blockIdx.x * blockDim.x + threadIdx.x;
    int local_thid = threadIdx.x;
    int offset = 1;
    int intsPerBlock = blockDim.x << 1;
    int intPosInBlock = local_thid << 1;

    if (thid < len) {
        temp[intPosInBlock] = d_Input[thid << 1]; // load input into shared memory
        temp[intPosInBlock + 1] = d_Input[(thid << 1) + 1];
    } else {
        temp[intPosInBlock] = 0; // load input into shared memory
        temp[intPosInBlock + 1] = 0;
    }

    for (int d = blockDim.x; d > 0; d >>= 1) { // build sum in place up the tree
        __syncthreads();
        if (local_thid < d) {
            int ai = offset * (intPosInBlock + 1) - 1;
            int bi = offset * (intPosInBlock + 2) - 1;

            temp[bi] += temp[ai];
        }
        offset <<= 1;
    }

    if (local_thid == 0) {
        temp[intsPerBlock - 1] = 0; // clear the last element
    }

    for (int d = 1; d < intsPerBlock; d <<= 1) { // traverse down tree & build scan
        offset >>= 1;
        __syncthreads();
        if (local_thid < d) {
            int ai = offset * (intPosInBlock + 1) - 1;
            int bi = offset * (intPosInBlock + 2) - 1;

            int t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }
    __syncthreads();

    d_Output[thid << 1] = temp[intPosInBlock]; // write results to device memory
    d_Output[(thid << 1) + 1] = temp[intPosInBlock + 1];
}

__global__ void blockScanNoConflict(int *d_Output, int *d_Input, int len) {
    int thid = blockIdx.x * blockDim.x + threadIdx.x;
    int local_thid = threadIdx.x;
    int offset = 1;
    int intsPerBlock = blockDim.x << 1;
    int intPosInBlock = local_thid << 1;

    int ai = local_thid;
    int bi = local_thid + (blockDim.x);
    int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
    int bankOffsetB = CONFLICT_FREE_OFFSET(bi);
    int outputAi = ai;
    int outputBi = bi;
    if (thid < len) {
        temp[ai + bankOffsetA] = d_Input[thid];
        temp[bi + bankOffsetB] = d_Input[thid + blockDim.x];
    } else {
        temp[ai + bankOffsetA] = 0;
        temp[bi + bankOffsetB] = 0;
    }

    for (int d = blockDim.x; d > 0; d >>= 1) { // build sum in place up the tree
        __syncthreads();
        if (local_thid < d) {
            ai = offset * (intPosInBlock + 1) - 1;
            bi = offset * (intPosInBlock + 2) - 1;
            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            temp[bi] += temp[ai];
        }
        offset <<= 1;
    }

    if (local_thid == 0) {
        temp[intsPerBlock - 1 + CONFLICT_FREE_OFFSET(intsPerBlock - 1)] = 0;
    }

    for (int d = 1; d < intsPerBlock; d <<= 1) { // traverse down tree & build scan
        offset >>= 1;
        __syncthreads();
        if (local_thid < d) {
            ai = offset * (intPosInBlock + 1) - 1;
            bi = offset * (intPosInBlock + 2) - 1;
            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);
            outputAi = offset * ((thid << 1) + 1) - 1;
            outputBi = offset * ((thid << 1) + 2) - 1;
            outputAi += CONFLICT_FREE_OFFSET(outputAi);
            outputBi += CONFLICT_FREE_OFFSET(outputBi);

            int t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }
    __syncthreads();

    d_Output[outputAi] = temp[ai + bankOffsetA];
    d_Output[outputBi] = temp[bi + bankOffsetB];
}

__global__ void level1(int *d_Output, int *d_Input, int len, int *d_SumOutput) {
    int thid = blockIdx.x * blockDim.x + threadIdx.x;
    int local_thid = threadIdx.x;
    int offset = 1;
    int intsPerBlock = blockDim.x << 1;
    int intPosInBlock = local_thid << 1;

    int ai = local_thid;
    int bi = local_thid + (blockDim.x);
    int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
    int bankOffsetB = CONFLICT_FREE_OFFSET(bi);
    int outputAi = ai;
    int outputBi = bi;
    if (thid < len) {
        temp[ai + bankOffsetA] = d_Input[thid];
        temp[bi + bankOffsetB] = d_Input[thid + blockDim.x];
    } else {
        temp[ai + bankOffsetA] = 0;
        temp[bi + bankOffsetB] = 0;
    }

    for (int d = blockDim.x; d > 0; d >>= 1) { // build sum in place up the tree
        __syncthreads();
        if (local_thid < d) {
            ai = offset * (intPosInBlock + 1) - 1;
            bi = offset * (intPosInBlock + 2) - 1;
            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            temp[bi] += temp[ai];
        }
        offset <<= 1;
    }

    if (local_thid == 0) {
        d_SumOutput[blockIdx.x] = temp[intsPerBlock - 1 + CONFLICT_FREE_OFFSET(intsPerBlock - 1)];
        temp[intsPerBlock - 1 + CONFLICT_FREE_OFFSET(intsPerBlock - 1)] = 0;
    }

    for (int d = 1; d < intsPerBlock; d <<= 1) { // traverse down tree & build scan
        offset >>= 1;
        __syncthreads();
        if (local_thid < d) {
            ai = offset * (intPosInBlock + 1) - 1;
            bi = offset * (intPosInBlock + 2) - 1;
            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);
            outputAi = offset * ((thid << 1) + 1) - 1;
            outputBi = offset * ((thid << 1) + 2) - 1;
            outputAi += CONFLICT_FREE_OFFSET(outputAi);
            outputBi += CONFLICT_FREE_OFFSET(outputBi);

            int t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }
    __syncthreads();

    d_Output[outputAi] = temp[ai + bankOffsetA];
    d_Output[outputBi] = temp[bi + bankOffsetB];
}

__global__ void addToBlocks(int *d_Output, int *d_IncrOutput) {
    int thid = blockIdx.x * blockDim.x + threadIdx.x;

    d_Output[thid] += d_IncrOutput[blockIdx.x];
}

void fullscan(int *d_Output, int *d_Input, int len, int blockSize) {
    int gridSize = (len + (blockSize * 2) - 1) / (blockSize * 2);
    int sharedMemSize = blockSize * 2;
    int *d_SumsOutput;
    int *d_IncrOutput;
    int *h_SumsOutput_d;
    int *h_IncrOutput_d;
    int *h_Output_d;

    h_Output_d = (int *) malloc(len * sizeof(int));
    h_SumsOutput_d = (int *) malloc(gridSize * sizeof(int));
    h_IncrOutput_d = (int *) malloc(gridSize * sizeof(int));
    h_Output_d = (int *) malloc(len * sizeof(int));

    checkCudaErrors(hipMalloc((void **) &d_SumsOutput, gridSize * sizeof(int)));
    checkCudaErrors(hipMalloc((void **) &d_IncrOutput, gridSize * sizeof(int)));

    level1 << < gridSize, blockSize, sharedMemSize >> > (d_Output, d_Input, len, d_SumsOutput);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipMemcpy(h_SumsOutput_d, d_SumsOutput, gridSize * sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_Output_d, d_Output, len * sizeof(int), hipMemcpyDeviceToHost));

    printf("Block Scan Result:\n");
    for (int i = 0; i < len; i++) {
        printf("%d ", h_Output_d[i]);
    }
    printf("\n");

    printf("SUM Array:\n");
    for (int i = 0; i < gridSize; i++) {
        printf("%d ", h_SumsOutput_d[i]);
    }
    printf("\n");

    blockScanNoConflict << < gridSize, blockSize * 2, sharedMemSize >> >
                                                      (d_IncrOutput, d_SumsOutput, gridSize * sizeof(int));
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipMemcpy(h_IncrOutput_d, d_IncrOutput, gridSize * sizeof(int), hipMemcpyDeviceToHost));
    printf("INCR Scan Result:\n");
    for (int i = 0; i < gridSize; i++) {
        printf("%d ", h_IncrOutput_d[i]);
    }
    printf("\n");

    printf("Grid: %d, Block %d\n", gridSize, blockSize);
    addToBlocks << < gridSize, blockSize * 2, sharedMemSize >> > (d_Output, d_IncrOutput);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipMemcpy(h_Output_d, d_Output, len * sizeof(int), hipMemcpyDeviceToHost));
    printf("Add to Blocks Result:\n");
    for (int i = 0; i < len; i++) {
        printf("%d ", h_Output_d[i]);
    }
    printf("\n");
}

void printTestEquals(int *value1, int *value2, int len) {
    bool equal = true;
    for (int i = 0; i < len; i++) {
        if (value1[i] != value2[i]) {
            equal = false;
        }
    }

    printf("Test: %s\n", equal ? "PASS" : "FAIL");
}

void printTestBlockEquals(int *complete, int *blockOnly, int len, int blockSize) {
    bool equal = true;
    for (int i = 0; i < len; i++) {
        if (complete[i % (blockSize * 2)] != blockOnly[i]) {
            equal = false;
        }
    }

    printf("Test: %s\n", equal ? "PASS" : "FAIL");
}

int main() {
    int len = 32; // Number of elements in the input array.
    int blockSize = 2;
    int gridSize = (len + (blockSize * 2) - 1) / (blockSize * 2);
    int sharedMemSize = 2 * blockSize * sizeof(int);
    int testCount = 5;

    double timerResult;

    int *h_Input;
    int *h_Output;
    int *h_Output_d;
    int *d_Input;
    int *d_Output;
    StopWatchInterface *timer = NULL;


    // Allocate host memory
    printf("Allocating host memory...\n");
    h_Input = (int *) malloc(len * sizeof(int));
    h_Output = (int *) malloc(len * sizeof(int));
    h_Output_d = (int *) malloc(len * sizeof(int));

    // Create timer
    sdkCreateTimer(&timer);

    // Generate random integers to create input
    printf("Generating random input...\n");
    srand((uint) (time(NULL)));
    for (int i = 0; i < len; i++) {
        h_Input[i] = 1; //rand() % 10;
    }

    // Allocate device memory and copy input to device
    printf("Allocating device memory...\n");
    checkCudaErrors(hipMalloc((void **) &d_Input, len * sizeof(int)));
    checkCudaErrors(hipMalloc((void **) &d_Output, len * sizeof(int)));
    printf("Copying input to device...\n");
    checkCudaErrors(hipMemcpy(d_Input, h_Input, len * sizeof(int), hipMemcpyHostToDevice));


    /**
     * Reference Scan
     */
    refScan(h_Output, h_Input, len);


    /**
     * Test Naive Block Scan
     */

    // Start timers
    sdkResetTimer(&timer);
    sdkStartTimer(&timer);

    // Run kernel
    for (int i = 0; i < testCount; i++) {
        blockScan << < gridSize, blockSize, sharedMemSize >> > (d_Output, d_Input, len);
    }
    checkCudaErrors(hipDeviceSynchronize());

    // Stop timer
    sdkStopTimer(&timer);

    // Print results
    checkCudaErrors(hipMemcpy(h_Output_d, d_Output, len * sizeof(int), hipMemcpyDeviceToHost));
    printf("\nBlock Scan Result:\n");
    printf("Grid Size: %d, Block Size: %d\n", gridSize, blockSize);
    printf("Block Scan Result:\n");
    printTestBlockEquals(h_Output, h_Output_d, len, blockSize);
    timerResult = 1.0e-3 * sdkGetTimerValue(&timer) / testCount;
    printf("Time taken: %.5f s, Number of Elements: %d\n\n", timerResult, len);


    /**
     * Test Block Scan with Bank Collision Avoidance Optimisation
     */

    // Reset memory
    h_Output_d = (int *) memset(h_Output_d, 0, len * sizeof(int));
    checkCudaErrors(hipMemset(d_Output, 0, len * sizeof(int)));

    // Start timers
    sdkResetTimer(&timer);
    sdkStartTimer(&timer);

    // Run kernel
    for (int i = 0; i < testCount; i++) {
        blockScanNoConflict << < gridSize, blockSize, sharedMemSize >> > (d_Output, d_Input, len);
    }
    checkCudaErrors(hipDeviceSynchronize());

    // Stop timer
    sdkStopTimer(&timer);

    // Print results
    checkCudaErrors(hipMemcpy(h_Output_d, d_Output, len * sizeof(int), hipMemcpyDeviceToHost));
    printf("\nBlock Scan With BCAO Result:\n");
    printf("Grid Size: %d, Block Size: %d\n", gridSize, blockSize);
    printTestBlockEquals(h_Output, h_Output_d, len, blockSize);
    timerResult = 1.0e-3 * sdkGetTimerValue(&timer) / testCount;
    printf("Time taken: %.5f s, Number of Elements: %d\n\n", timerResult, len);


    /**
     * Test Full Scan
     */
    h_Output_d = (int *) memset(h_Output_d, 0, len * sizeof(int));
    checkCudaErrors(hipMemset(d_Output, 0, len * sizeof(int)));
    fullscan(d_Output, d_Input, len, blockSize);

    // Clean up memory
    printf("Cleaning up memory...\n");
    free(h_Input);
    free(h_Output_d);
    checkCudaErrors(hipFree(d_Output));
    checkCudaErrors(hipFree(d_Input));
}