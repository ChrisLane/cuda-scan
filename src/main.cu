#include "hip/hip_runtime.h"
/**
 * == Personal ==
 * Name: Christopher Lane
 * ID: 1435876
 *
 * == Goals ==
 * Block Scan: Achieved
 * Full Scan: Not Achieved
 * BCAO: Achieved
 *
 * == Times ==
 ∗ Block scan w/o BCAO: 0.00199 s
 ∗ Block scan w/ BCAO:  0.00214 s
 ∗ Full scan w/o BCAO:
 ∗ Full scan w/ BCAO:
 *
 * == Hardware ==
 * CPU: i7-4710MQ
 * GPU: GTX 860M
 *
 * == Comments ==
 * Implementation:
 * - Length of array is padded to the nearest power of 2 and the array is padded with 0s to that length.
 *
 * Performance:
 *
 */

#include <stdio.h>
#include <math.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#define NUM_BANKS 32
#define LOG_NUM_BANKS 5
#define CONFLICT_FREE_OFFSET(n) \
    ((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))

void refScan(int *h_output, int *h_input, const int len) {
    h_output[0] = 0;

    for (int i = 1; i < len; ++i) {
        h_output[i] = h_input[i - 1] + h_output[i - 1];
    }
}

extern __shared__ int temp[]; // allocated on invocation
__global__ void blockScan(int *d_Output, int *d_Input, int len) {
    int thid = blockIdx.x * blockDim.x + threadIdx.x;
    int local_thid = threadIdx.x;
    int offset = 1;
    int intsPerBlock = blockDim.x << 1;
    int intPosInBlock = local_thid << 1;

    if (thid < len) {
        temp[intPosInBlock] = d_Input[thid << 1]; // load input into shared memory
        temp[intPosInBlock + 1] = d_Input[(thid << 1) + 1];
    } else {
        temp[intPosInBlock] = 0; // load input into shared memory
        temp[intPosInBlock + 1] = 0;
    }

    for (int d = blockDim.x; d > 0; d >>= 1) { // build sum in place up the tree
        __syncthreads();
        if (local_thid < d) {
            int ai = offset * (intPosInBlock + 1) - 1;
            int bi = offset * (intPosInBlock + 2) - 1;

            temp[bi] += temp[ai];
        }
        offset <<= 1;
    }

    if (local_thid == 0) {
        temp[intsPerBlock - 1] = 0; // clear the last element
    }

    for (int d = 1; d < intsPerBlock; d <<= 1) { // traverse down tree & build scan
        offset >>= 1;
        __syncthreads();
        if (local_thid < d) {
            int ai = offset * (intPosInBlock + 1) - 1;
            int bi = offset * (intPosInBlock + 2) - 1;

            int t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }
    __syncthreads();

    d_Output[thid << 1] = temp[intPosInBlock]; // write results to device memory
    d_Output[(thid << 1) + 1] = temp[intPosInBlock + 1];
}

__global__ void blockScanNoConflict(int *d_Output, int *d_Input, int len) {
    int thid = blockIdx.x * blockDim.x + threadIdx.x;
    int local_thid = threadIdx.x;
    int offset = 1;
    int intsPerBlock = blockDim.x << 1;
    int intPosInBlock = local_thid << 1;

    int ai = local_thid;
    int bi = local_thid + (blockDim.x);
    int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
    int bankOffsetB = CONFLICT_FREE_OFFSET(bi);
    if (thid < len) {
        temp[ai + bankOffsetA] = d_Input[thid];
        temp[bi + bankOffsetB] = d_Input[thid + blockDim.x];
    } else {
        temp[ai + bankOffsetA] = 0;
        temp[bi + bankOffsetB] = 0;
    }

    for (int d = blockDim.x; d > 0; d >>= 1) { // build sum in place up the tree
        __syncthreads();
        if (local_thid < d) {
            ai = offset * (intPosInBlock + 1) - 1;
            bi = offset * (intPosInBlock + 2) - 1;
            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            temp[bi] += temp[ai];
        }
        offset <<= 1;
    }

    if (local_thid == 0) {
        temp[intsPerBlock - 1 + CONFLICT_FREE_OFFSET(intsPerBlock - 1)] = 0;
    }

    for (int d = 1; d < intsPerBlock; d <<= 1) { // traverse down tree & build scan
        offset >>= 1;
        __syncthreads();
        if (local_thid < d) {
            ai = offset * (intPosInBlock + 1) - 1;
            bi = offset * (intPosInBlock + 2) - 1;
            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            int t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }
    __syncthreads();

    d_Output[ai] = temp[ai + bankOffsetA];
    d_Output[bi] = temp[bi + bankOffsetB];
}

void printTestEquals(int *value1, int *value2, int len) {
    bool equal = true;
    for (int i = 0; i < len; i++) {
        if (value1[i] != value2[i]) {
            equal = false;
        }
    }

    printf("Test: %s\n", equal ? "PASS" : "FAIL");
}

void printTestBlockEquals(int *complete, int *blockOnly, int len, int blockSize) {
    bool equal = true;
    for (int i = 0; i < len; i++) {
        if (complete[i % (blockSize * 2)] != blockOnly[i]) {
            equal = false;
        }
    }

    printf("Test: %s\n", equal ? "PASS" : "FAIL");
}

int main() {
    int len = 10000000; // Number of elements in the input array.
    int blockSize = 128;
    int gridSize = (len + (blockSize * 2) - 1) / (blockSize * 2);
    int sharedMemSize = 2 * blockSize * sizeof(int);
    int testCount = 5;

    double timerResult;

    int *h_Input;
    int *h_Output;
    int *h_Output_d;
    int *d_Input;
    int *d_Output;
    StopWatchInterface *timer = NULL;


    // Allocate host memory
    printf("Allocating host memory...\n");
    h_Input = (int *) malloc(len * sizeof(int));
    h_Output = (int *) malloc(len * sizeof(int));
    h_Output_d = (int *) malloc(len * sizeof(int));

    // Create timer
    sdkCreateTimer(&timer);

    // Generate random integers to create input
    printf("Generating random input...\n");
    srand((uint) (time(NULL)));
    for (int i = 0; i < len; i++) {
        h_Input[i] = 1; //rand() % 10;
    }

    // Allocate device memory and copy input to device
    printf("Allocating device memory...\n");
    checkCudaErrors(hipMalloc((void **) &d_Input, len * sizeof(int)));
    checkCudaErrors(hipMalloc((void **) &d_Output, len * sizeof(int)));
    printf("Copying input to device...\n");
    checkCudaErrors(hipMemcpy(d_Input, h_Input, len * sizeof(int), hipMemcpyHostToDevice));

    // Run reference scan
    refScan(h_Output, h_Input, len);

    // Run Single block scan
    sdkResetTimer(&timer);
    sdkStartTimer(&timer);
    for (int i = 0; i < testCount; i++) {
        blockScan << < gridSize, blockSize, sharedMemSize >> > (d_Output, d_Input, len);
    }
    checkCudaErrors(hipDeviceSynchronize());
    sdkStopTimer(&timer);

    checkCudaErrors(hipMemcpy(h_Output_d, d_Output, len * sizeof(int), hipMemcpyDeviceToHost));
    printf("\nBlock Scan Result:\n");
    printf("Grid Size: %d, Block Size: %d\n", gridSize, blockSize);
    printf("Block Scan Result:\n");
    printTestBlockEquals(h_Output, h_Output_d, len, blockSize);
    timerResult = 1.0e-3 * sdkGetTimerValue(&timer) / testCount;
    printf("Time taken: %.5f s, Number of Elements: %d\n\n", timerResult, len);

    // Run block scan no bank conflicts
    sdkResetTimer(&timer);
    sdkStartTimer(&timer);
    for (int i = 0; i < testCount; i++) {
        blockScanNoConflict << < gridSize, blockSize, sharedMemSize >> > (d_Output, d_Input, len);
    }
    checkCudaErrors(hipDeviceSynchronize());
    sdkStopTimer(&timer);

    checkCudaErrors(hipMemcpy(h_Output_d, d_Output, len * sizeof(int), hipMemcpyDeviceToHost));
    printf("\nBlock Scan With BCAO Result:\n");
    printf("Grid Size: %d, Block Size: %d\n", gridSize, blockSize);
    printTestBlockEquals(h_Output, h_Output_d, len, blockSize);
    timerResult = 1.0e-3 * sdkGetTimerValue(&timer) / testCount;
    printf("Time taken: %.5f s, Number of Elements: %d\n\n", timerResult, len);

    // Clean up memory
    printf("Cleaning up memory...\n");
    free(h_Input);
    free(h_Output_d);
    checkCudaErrors(hipFree(d_Output));
    checkCudaErrors(hipFree(d_Input));
}